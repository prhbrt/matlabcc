#include "hip/hip_runtime.h"
#define MATLABCC_REPORT_STACK

#include <cmath>
#include <iostream>
#include <fstream>

#include <matlabcc>
#include <iostream>
#include <stdio.h>

static void HandleError(
  hipError_t err,
  const char *file,
  int line )
{
  if (err != hipSuccess)
  {
    printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
    exit( EXIT_FAILURE );
  }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


#define HANDLE_NULL( a ) \\
{ \\
  if (a == NULL)\\
  { \\
    printf( "Host memory failed in %s at line %d\n", __FILE__, __LINE__ ); \\
    exit( EXIT_FAILURE ); \\
  } \\
}


#define N 1400
#define K 15
#define K2 ((K - 1) / 2)


template<int mx, int my>
__device__ inline int index(int x, int y)
{
  return x*my + y;
}

__global__ void ssimkernel(double *test, double *reference, double *kernel, double *ssim)
{
  int x = blockIdx.x;
  int y = blockIdx.y;
  
  double mean_test = 0;
  double mean_reference = 0;
  for(int i=0; i<K; ++i)
  for(int j=0; j<K; ++j)
  {
    if (x + i - K2 >= 0 and
        x + i - K2 < N  and
        y + j - K2 >= 0 and
        y + j - K2 < N)
    {
      mean_test      +=  kernel[index<K,K>(i,j)] * test[index<N,N>(x+i-K2, y+j-K2)];
      mean_reference +=  kernel[index<K,K>(i,j)] * reference[index<N,N>(x+i-K2, y+j-K2)];
    }
  }
  
  double var_test = 0;
  double var_reference = 0;
  double correlation = 0;
  
  for(int i=0; i<K; ++i)
  for(int j=0; j<K; ++j)
  {
    if (x + i - K2 >= 0 and
        x + i - K2 < N  and
        y + j - K2 >= 0 and
        y + j - K2 < N)
    {
      double a = (test[index<N,N>(x+i-K2, y+j-K2)] - mean_test);
      double b = (reference[index<N,N>(x+i-K2, y+j-K2)] - mean_reference);
      var_test      += kernel[index<K,K>(i,j)] * a * a;
      var_reference += kernel[index<K,K>(i,j)] * b * b;
      correlation   += kernel[index<K,K>(i,j)] * a * b;
    }
  }
}


template<typename T>
inline T sim(T v0, T v1, T c)
{
  return (c + 2*v0*v1) / (c + v1*v1 + v0*v0);
}

void mexFunction(int nargout, mxArray *argout[], int nargin, const mxArray *argin[])
{
  mtb::Mat<double> im ();
  mtb::Mat<double> imr(argin[1]);
  mtb::Mat<double> kernel(argin[2]);

  double *test      = (double *)mxGetData(argin[0]);
  double *reference = (double *)mxGetData(argin[1]);
  double *kernel    = (double *)mxGetData(argin[2]);
  
  mxSize rows = 
  mxSize cols = 
  
  
  double *kernel    = (double *)mxGetData(argin[2]);
  double *ssim      = new double[N * N];
  
  double *test_cuda;
  double *reference_cuda;
  double *kernel_cuda;
  double *ssim_cuda;
  HANDLE_ERROR( hipMalloc((void **) &test_cuda,      N*N*sizeof(double)) );
  HANDLE_ERROR( hipMalloc((void **) &reference_cuda, N*N*sizeof(double)) );
  HANDLE_ERROR( hipMalloc((void **) &kernel_cuda,    K*K*sizeof(double)) );
  HANDLE_ERROR( hipMalloc((void **) &ssim_cuda,      N*N*sizeof(double)) );
  
  HANDLE_ERROR( hipMemcpy(test_cuda,      test,      N*N*sizeof(double), hipMemcpyHostToDevice) );
  HANDLE_ERROR( hipMemcpy(reference_cuda, reference, N*N*sizeof(double), hipMemcpyHostToDevice) );
  HANDLE_ERROR( hipMemcpy(kernel_cuda,    kernel,    K*K*sizeof(double), hipMemcpyHostToDevice) );
  
  dim3 grid   ( N, N );
  ssimkernel<<<grid, 1>>>(test_cuda, reference_cuda, kernel_cuda, ssim_cuda);
  
  hipFree( test_cuda );
  hipFree( reference_cuda );
  hipFree( kernel_cuda );
  hipFree( ssim_cuda );
  
  delete[] test;
  delete[] reference;
  delete[] kernel;
  delete[] ssim;
  
  argout[0] = mtb::create<3, double>(im.size(0), im.size(1), im.size(2));
  mtb::Mat<double, false> ssim(argout[0]);
  
  size_t kr = kernel.size(0) / 2;
  size_t kc = kernel.size(1) / 2;
  
  mtb::forEachElement<3>(im, [&](size_t row, size_t col, size_t channel) {
    double mean = 0, meanr = 0;
    double sum = 0;;
    mtb::forKernel(im, row, col, kernel, kr, kc, [&](double kv, size_t row, size_t col) {
      mean  += kv * im (row, col, channel);
      meanr += kv * imr(row, col, channel);
      sum += kv;
    });
    mean  /= sum;
    meanr /= sum;
    
    double var = 0, varr = 0, cov = 0;
    mtb::forKernel(im, row, col, kernel, kr, kc, [&](double kv, size_t row, size_t col) {
      var  += kv * (im (row, col, channel) - mean ) * (im (row, col, channel) - mean );
      varr += kv * (imr(row, col, channel) - meanr) * (imr(row, col, channel) - meanr);
      cov  += kv * (im (row, col, channel) - mean ) * (imr(row, col, channel) - meanr);
    });
    var  /= sum;
    varr /= sum;
    cov  /= sum;
    //var = std::sqrt(var);
    //varr = std::sqrt(varr);
    
    ssim(row, col, channel) = sim(mean, meanr, 0.01) * (0.03 + 2*cov) / (0.03 + var + varr);
  });
}
